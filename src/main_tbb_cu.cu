#include <algorithm>
#include <iostream>
#include <numeric>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>

#define MET_IS_CUDA_ENABLED

#ifdef MET_IS_CUDA_ENABLED
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#define MET_GPU __device__
#define MET_CPU_GPU __host__ __device__
#endif

#ifndef MET_IS_CUDA_ENABLED
#define MET_GPU
#define MET_CPU_GPU
#include <tbb/blocked_range.h>
#include <tbb/parallel_for.h>
#include <tbb/parallel_reduce.h>
#endif

using uint = unsigned int;



template <typename F>
void parallel_for(uint n_items, F function) {
#ifdef MET_IS_CUDA_ENABLED
  // Launch CUDA GPU kernel
  auto kernel = &CUDAKernel<F>; // TODO: are these cached???
  uint block_size = 256; // TODO: determine optimal block size on the fly
  uint grid_size = (n_items + block_size - 1) / block_size;
  kernel<<<grid_size, block_size>>>(n_items, function);
#else
  // Launch TBB operation instead
  tbb::parallel_for<size_t>(0, n_items, function);
  // tbb::parallel_for<size_t>(0, n_items, [&](auto range) {
  //   function(range);
  // });
#endif
}

/* template <typename Value, typename F>
Value parallel_reduce(uint n_items, Value base, F function) {
#ifdef MET_IS_CUDA_ENABLED
  
#else
  return tbb::parallel_reduce<size_t>(0, n_items, base, function);
  // return tbb::parallel_reduce<size_t>(0, n_items, base, [=](tbb::blocked_range<int> r, Value t) {
  //   for (int i = r.begin(); i < r.end(); ++i) {
  //     function(t);
  //   }
  // });
#endif
} */



int main(int argc, char** argv) {
  uint n_items = 32;
  thrust::host_vector<uint> v(n_items);
  std::iota(v.begin(), v.end(), 0);
  thrust::device_vector<uint> v_ = v;

  thrust::for_each(v_.begin(), v_.end(), [] MET_CPU_GPU (uint &i) {
    printf("%i\n", i);
  });

  auto v_ptr = thrust::raw_pointer_cast(v_.data());
  parallel_for(v_.size(), [=] MET_CPU_GPU (uint &i) mutable {
    printf("%i\n", v_ptr[i]);
  });

  // for (auto& i : v) {
  //   printf("%i\n", v_[i]);
  // }




  // std::vector<uint> v(n_items);

  // parallel_for(n_items, [&] MET_CPU_GPU (uint i) {
  //   printf("%i\n", v[i]);
  // });

  // uint r = parallel_reduce(n_items, 0, [&] MET_CPU_GPU (uint i, uint total) {
  //   total += v[i];
  // });
  
  // printf("total: %i\n", r);

  return 0;
}